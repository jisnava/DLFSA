
#include <hip/hip_runtime.h>
#include<string>
#include<fstream>
#include<iostream>
#include<vector>
#include<stdio.h>
#include<algorithm>
using namespace std;
typedef struct cord3d
{
	float x;
	float y;
	float z;
}cord3d;

void fileprop(string, int *, int *, char**);
void setcords(cord3d *, int *, string);
void write_to_file(int *, cord3d *, char*, int, int, string);

__device__ bool isspace(char c)
{
	if(c==' ' || c=='\t' || c=='\n' || c=='\v' || c=='\f' || c=='\r')
		return true;
	return false;
}

__device__ bool isupper(char c)
{
	if(c>='A' && c<='Z')
		return true;
	return false;
}

__device__ bool isdigit(char c)
{
	if(c>='0' && c<='9')
		return true;
	return false;
}

//cords is a pair of 3 numbers in the form ( x1x2, y1y2, z1z2 ) where x1 and x2 are the line offset of start and end of x coordinate 
__global__ void extract_cords(char * file, int lc, int ls, int * cords, bool line_num) 
{

	if( file[(blockIdx.x)*ls]=='A' && file[(blockIdx.x)*ls+1]=='T' && file[(blockIdx.x)*ls+2]=='O' && file[(blockIdx.x)*ls+3]=='M' )
	{	
	int state=0; //state of automata
	int cordno=-1; // the coordinate number that you are currently on		

	for(int i=((blockIdx.x)*ls)+4 ; i<((blockIdx.x)*ls)+ls ; )
        {
	// Atom to state 1, branch 1
		if(state==0 && isspace(file[i]))
		{
			int j=0;
			while(isspace(file[i]) && j<6)
			{
				i++;
				j++;
			}
			state=1;
			continue;
		}
	// Atom to state 2, branch 2; state 1 to state 2 branch 2;
		else if((state==0 || state==1) && isdigit(file[i]))
		{
			if(state==0)
				printf("No space found after ATOM on line %d \n", blockIdx.x);
			int j=0;
			while(isdigit(file[i]) && j<7)
			{
				i++;
				j++;
			}
			state=2;
		}
	// Atom to dead state , branch 3
		else if(state==0 && (!isspace(file[i]) && !isupper(file[i])))
		{
			printf("3rd Branch of state 0 with block id as %d\n",blockIdx.x);
			break;	
		}
	// State 1 to dead state, branch 1; state 10 to dead state, branch 1
		else if((state==1 || state==10) && !isdigit(file[i]))
		{
			printf("1st Branch from state 1 or 10\n");
			break;
		}
	// State 2 to 3 branch 2
		else if(state==2 && isspace(file[i]))
		{
			int j=0;
			while(isspace(file[i]) && j<9)
			{
				i++;
				j++;
			}
			state=3;
		}
	// State 2 to dead state, branch 1; state 5 to dead state branch 1
		else if((state==2 || state==5) && isdigit(file[i]))
		{
			if(state==2)
			printf("State 2 to dead state, branch 1 standing at %c\n",file[i]);
			else
			printf("State 5 to dead state, branch 1\n");
			break;
		}
	// State 2 to state 4, branch 3; state 3 to state 4 branch 2
		else if((state==2 || state==3 ) && isupper(file[i]))
		{
			if(state==2)
				printf("No space found after,State 2 to state 4, branch 3\n");
			int j=0;
			while( isupper(file[i]) && j<4 )
			{
				i++;
				j++;
			}
			state=4;
		}
	// State 3 to dead state, branch 1 ; State 6 to dead state, branch 1; state 8 to dead state, branch 1
		else if((state==3 || state==6) && !isupper(file[i]))
		{
			printf("State 3 or 6 or 8 to dead state, branch 1\n");
			break;
		}
	// State 4 to 5, branch 2
		else if(state==4 && isdigit(file[i]))
		{
			i++;
			state=5;
		}
	// State 4 to 6, branch 1; state 5 to 6 branch 2
		else if((state==4 || state==5) && isspace(file[i]))
		{
			int j=0;
			while(isspace(file[i]) && j<6)
			{
				i++;
				j++;
			}
			state=6;
		}
	// State 4 to 7, branch 3; State 5 to 7 branch 3; state 6 to 7 branch 2
		else if((state==4 || state==5 || state==6) && isupper(file[i]))
		{
			if(state==4 || state ==5)
				printf("State 4 or 5 to 7, branch 3\n ");
			int j=0;
			while(isupper(file[i]) && j<3)
			{
				i++;
				j++;
			}
			state=7;
		}
	// State 7 to 8, branch 2
		else if(state==7 && isspace(file[i]))
		{
			i++;
			state=8;
		}
	// State 7 to dead state; state 9 to dead state; state 11 to dead state
		else if( (state==7 || state==9 || state==11) && !isspace(file[i]))
		{
			printf("State 7 or 9 or 11 to dead state\n");
			break;
		}
	// State 8 to 9
		else if( state==8 && isupper(file[i]))
		{
			i++;
			state=9;
		}
	// State 9 to 10
		else if( state==9 && isspace(file[i]))
		{
			int j=0;
			while(isspace(file[i]) && j<4)
			{
				j++;
				i++;
			}
			state=10;
		}
	// State 10 to 11
		else if( state==10 && isdigit(file[i]))
		{
			int j=0;
			while(isdigit(file[i]) && j<8)
			{
				j++;
				i++;
			}
			state=11;
		}
	// State 11 to 12
		else if( state==11 && isspace(file[i]))
		{
			int j=0;
			while(isspace(file[i]) && j<19)
			{
				j++;
				i++;	
			}
			state=12;
		}

	// From this pint onwards we are storing the value of the coordinates
		else if( state==12 && ( file[i]=='-' || isdigit(file[i]) ) && cordno < 2)
		{
			cordno++;
			cords[(blockIdx.x)*6 + cordno*2]=(i-(blockIdx.x)*ls);
			int j=0;
			i++;
			while((isdigit(file[i]) || file[i]=='.') && j<19)
			{
				j++;
				i++;
			}
			cords[(blockIdx.x)*6+1 + cordno*2]=(i-(blockIdx.x)*ls-1);
		}

		else if( state==12 && isspace(file[i]) && cordno < 3)
		{
			int j=0;
			while(isspace(file[i]) && j<10)
			{
				j++;
				i++;
			}
		}

		else
			break;
	}

	}

}
int main(int arg_count, char** arg_vector) // Arguments : filename 
{
	int ls;
	int lc;
	char *file; 
	
	if(arg_count==2)
		fileprop(arg_vector[1], &ls, &lc, &file);  //Initializes the file in file, have a more thoughtful name
	else
	{	cout<<"arguments wrong, exiting\n";
		exit(0);
	}

	int cords[2*3*lc]; 
	bool line_num[lc]; // Line numbers which have corrdinates, [clean up, didn't use]
	cord3d mycords[lc]; // 3dcords is a structure

	int * cords_d;
	char * file_d;
	bool * line_num_d;

	hipMalloc((void**)&cords_d, 2*3*lc * sizeof(int)); 
	hipMalloc((void**)&file_d, (lc*ls+lc)*sizeof(char));
	hipMalloc((void**)&line_num_d, (lc)*sizeof(bool));

	hipMemcpy(file_d, file, (lc*ls+lc)*sizeof(char), hipMemcpyHostToDevice);

	extract_cords<<<lc,1>>>(file_d, lc, ls, cords_d, line_num_d);
	hipMemcpy(cords, cords_d, 2*3*lc * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(line_num, line_num_d, lc * sizeof(bool), hipMemcpyDeviceToHost);

	setcords( mycords, cords, arg_vector[1] ); // arg_vector[1] is the file name, may be make a separate variable for it

// Set the coordinates to offset it to zero, taking first one as the starting point
	float x = mycords[0].x;
	float y = mycords[0].y;
	float z = mycords[0].z;

	for(int i=0;i<lc;i++)
	{
		mycords[i].x = mycords[i].x - x;
		mycords[i].y = mycords[i].y - y;
		mycords[i].z = mycords[i].z - z;
	}

// Write it out to the file
	write_to_file(cords, mycords, file, lc, ls, arg_vector[1]); // cords in the pair of two values of positions

return 0;
}

void write_to_file(int * cords, cord3d * mycords, char * file, int lc, int ls, string filename)
{
	for(int i =0; i<lc; i++)
	{
		for(int f=0;f<3;f++)
		{
		float myucords;
		int g_i;
		if(f==0)
		{
		myucords=mycords[i].x; // my universal coords
		g_i=0; // global iterator for coords, i.e 0,2,4
		}
		else if(f==1)
		{
		myucords=mycords[i].y; // the universal coords
		g_i=2; // global iterator for coords, i.e 0,2,4
		}
		else
		{
		myucords=mycords[i].z; // the universal coords
		g_i=4; // global iterator for coords, i.e 0,2,4
		}
		
		if(file[i*ls + cords[6*i+g_i]]=='-')
		{
			if(myucords<0)
			{
				string val = to_string(myucords);
				char temp[10];
				int i_temp=0; //iterator for temp
				int i_dot=-1; //position of dot, set only if you have to work on it
				int len = val.length();
				temp[i_temp++]='-';
				int s_dot1=0; // saw dot in the first number? ( already present number )
				int s_dot2=0; // saw dot in the second number? ( number to be placed )
				for(int j=1,k=1; j<= cords[6*i+g_i+1]-cords[6*i+g_i] ; )
				{
					if(val[k]=='.') s_dot2=1;
				
					if(file[i*ls + cords[6*i+g_i]+j]=='.') 
					{
						s_dot1=1;
						if(j!=k)
							i_dot=j;
					}

					if( s_dot1 && !s_dot2 )
					{
						i_dot=j;
						temp[i_temp++]=val[k];
						k++;
						continue;
					}

					else if( s_dot2 && !s_dot1 )
					{
						for(int h=i_temp-1;h>=0;h--)
						{
							temp[h+1]=temp[h];
						}
						temp[0]=' ';
						i_temp++;
						j++;
					}
					else
					{
						if(k>=len)
						{
							file[i*ls + cords[6*i+g_i]+j]=' ';
							continue;
						}
	
						file[i*ls + cords[6*i+g_i]+j]=val[k];
						// Adding chars to the temp array if you have not encountered dot
						if(i_dot<0)
						{
							temp[i_temp++]=val[k];
						}
						// Done adding
						k++;
						j++;
					}
				}
				i_temp--;
				if(i_dot>0)
				{
					for(int j=i_dot-1;i_temp>=0;j--,i_temp--)
					{
						file[i*ls + cords[6*i+g_i]+j]=temp[i_temp];
					}
				}
			}
			else
			{
				string val = to_string(myucords);
				char temp[10];
				int i_temp=0; //iterator for temp
				int i_dot=-1; //position of dot, set only if you have to work on it
				int len = val.length();
				temp[i_temp++]=' ';
				file[i*ls + cords[6*i+g_i]]=' ';
				int s_dot1=0; // saw dot in the first number? ( already present number )
				int s_dot2=0; // saw dot in the second number? ( number to be placed )
				for(int j=1,k=0; j<= cords[6*i+1+g_i]-cords[6*i+g_i] ; )
				{
					if(val[k]=='.') s_dot2=1;
				
					if(file[i*ls + cords[6*i+g_i]+j]=='.') 
					{
						s_dot1=1;
						if((j-1)!=k)
							i_dot=j;
					}

					if( s_dot1 && !s_dot2 )
					{
						i_dot=j;
						temp[i_temp++]=val[k];
						k++;
						continue;
					}

					else if( s_dot2 && !s_dot1 )
					{
						for(int h=i_temp-1;h>=0;h--)
						{
							temp[h+1]=temp[h];
						}
						temp[0]=' ';
						i_temp++;
						j++;
					}
					else
					{
						if(k>=len)
						{
							file[i*ls + cords[6*i+g_i]+j]=' ';
							continue;
						}
	
						file[i*ls + cords[6*i+g_i]+j]=val[k];
						// Adding chars to the temp array if you have not encountered dot
						if(i_dot<0)
						{
							temp[i_temp++]=val[k];
						}
						// Done adding
						k++;
						j++;
					}
				}
				i_temp--;
				if(i_dot>0)
				{
					for(int j=i_dot-1;i_temp>=0;j--,i_temp--)
					{
						file[i*ls + cords[6*i+g_i]+j]=temp[i_temp];
					}
				}
			}
		}
		else
		{
			if(myucords<0)
			{
				string val = to_string(myucords);
				char temp[10];
				int i_temp=0; //iterator for temp
				int i_dot=-1; //position of dot, set only if you have to work on it
				int len = val.length();
				temp[i_temp++]='-';
				file[i*ls + cords[6*i+g_i]-1]='-';
				int s_dot1=0; // saw dot in the first number? ( already present number )
				int s_dot2=0; // saw dot in the second number? ( number to be placed )
				for(int j=0,k=1; j<=cords[6*i+1+g_i]-cords[6*i+g_i] ; )
				{
					if(val[k]=='.') s_dot2=1;
				
					if(file[i*ls + cords[6*i+g_i]+j]=='.') 
					{
						s_dot1=1;
						if(j!=(k-1))
							i_dot=j;
					}

					if( s_dot1 && !s_dot2 )
					{
						i_dot=j;
						temp[i_temp++]=val[k];
						k++;
						continue;
					}

					else if( s_dot2 && !s_dot1 )
					{
						for(int h=i_temp-1;h>=0;h--)
						{
							temp[h+1]=temp[h];
						}
						temp[0]=' ';
						i_temp++;
						j++;
					}
					else
					{
						if(k>=len)
						{
							file[i*ls + cords[6*i+g_i]+j]=' ';
							continue;
						}
	
						file[i*ls + cords[6*i+g_i]+j]=val[k];
						// Adding chars to the temp array if you have not encountered dot
						if(i_dot<0)
						{
							temp[i_temp++]=val[k];
						}
						// Done adding
						k++;
						j++;
					}
				}
				i_temp--;
				if(i_dot>0)
				{
					for(int j=i_dot-1;i_temp>=0;j--,i_temp--)
					{
						file[i*ls + cords[6*i+g_i]+j]=temp[i_temp];
					}
				}
			}
			else
			{

				string val = to_string(myucords);
				char temp[10];
				int i_temp=0; //iterator for temp
				int i_dot=-1; //position of dot, set only if you have to work on it
				int len = val.length();

				int s_dot1=0; // saw dot in the first number? ( already present number )
				int s_dot2=0; // saw dot in the second number? ( number to be placed )

				for(int j=0,k=0; j<= cords[6*i+1+g_i]-cords[6*i+g_i] ;  )
				{

					if(val[k]=='.') s_dot2=1;
				
					if(file[i*ls + cords[6*i+g_i]+j]=='.') 
					{
						s_dot1=1;
						if(j!=k)
							i_dot=j;
					}

					if( s_dot1 && !s_dot2 )
					{
						i_dot=j;
						temp[i_temp++]=val[k];
						k++;
						continue;
					}

					else if( s_dot2 && !s_dot1 )
					{
						for(int h=i_temp-1;h>=0;h--)
						{
							temp[h+1]=temp[h];
						}
						temp[0]=' ';
						i_temp++;
						j++;
					}

					else
					{
						if(k>=len)
						{
							file[i*ls + cords[6*i+g_i]+j]=' ';
							continue;
						}

						file[i*ls + cords[6*i+g_i]+j]=val[k];
						// Adding chars to the temp array if you have not encountered dot
						if(i_dot<0)
						{
							temp[i_temp++]=val[k];
						}
						// Done adding
						k++;
						j++;
					}
				

				}
				
				i_temp--;
				if(i_dot>0)
				{
					for(int j=i_dot-1;i_temp>=0;j--,i_temp--)
					{
						file[i*ls + cords[6*i+g_i]+j]=temp[i_temp];
					}
				}
			}
		}

	}
}
	ofstream outfile ("/mnt/pspdata/.init/sample-helices-DB/processed/"+filename+"_off0");
	for(int i=0; i<lc*ls;i++)
	{
		outfile<<file[i];
	}
	

}

void setcords( cord3d *mycords, int *cords, string filename) // cords is actually the start and stop offset in line of every xyz coords
{
	ifstream myfile (filename);
	int i=0;
	string line;
	while(getline(myfile, line))
	{
		mycords[i].x = stof(line.substr(cords[6*i],cords[6*i+1]-cords[6*i]+1));
		mycords[i].y = stof(line.substr(cords[6*i+2],cords[6*i+2+1]-cords[6*i+2]+1));
		mycords[i].z = stof(line.substr(cords[6*i+4],cords[6*i+4+1]-cords[6*i+4]+1));
		i++;
	}
}

void fileprop(string filename,int * ls, int * lc, char **file)
{
	
        ifstream myfile (filename);
	*lc = count(std::istreambuf_iterator<char>(myfile),std::istreambuf_iterator<char>(),'\n'); //linecount is number of lines

	myfile.clear();
	myfile.seekg(0, std::ios::beg);
	
	string line;    
        getline (myfile,line);        //Problem - Linesize may vary as we go down the file 
        *ls = line.size()+1;

	myfile.clear();
	myfile.seekg(0, std::ios::beg);

	*file=(char *)(malloc((*lc)*(*ls)*sizeof(char)));
        int it=0;
        char c;
        while (myfile.get(c))   
        {    
                (*file)[it]=c;    
                it++;
        }
}
