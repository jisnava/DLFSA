
#include <hip/hip_runtime.h>
#include<string>
#include<fstream>
#include<iostream>
#include<algorithm>
#include<vector>
#include<thread>
#include<mutex>
#include<condition_variable>
#include<queue>
#include<functional>
#include<chrono>
#include<pthread.h>
using namespace std;
int frag_size=5;
void process(string);

//--------------------------------------------------------------------------My threads
class ThreadPool
{
    public:

    ThreadPool (int threads) : shutdown_ (false)
    {
        // Create the specified number of threads
        threads_.reserve (threads);
        for (int i = 0; i < threads; ++i)
            threads_.emplace_back (std::bind (&ThreadPool::threadEntry, this, i));
    }

    ~ThreadPool ()
    {
        {
            // Unblock any threads and tell them to stop
            std::unique_lock <std::mutex> l (lock_);

            shutdown_ = true;
            condVar_.notify_all();
        }

        // Wait for all threads to stop
        // std::cerr << "Joining threads" << std::endl;
        for (auto& thread : threads_)
            thread.join();
    }

    void doJob (std::function <void (void)> func)
    {
        // Place a job on the queu and unblock a thread
        std::unique_lock <std::mutex> l (lock_);

        jobs_.emplace (std::move (func));
        condVar_.notify_one();
    }

    protected:

    void threadEntry (int i)
    {
        std::function <void (void)> job;

        while (1)
        {
            {
                std::unique_lock <std::mutex> l (lock_);

                while (! shutdown_ && jobs_.empty())
                    condVar_.wait (l);

                if (jobs_.empty ())
                {
                    // No jobs to do and we are shutting down
                    // std::cerr << "Thread " << i << " terminates" << std::endl;
                    return;
                 }

                // std::cerr << "Thread " << i << " does a job" << std::endl;
                job = std::move (jobs_.front ());
                jobs_.pop();
            }

            // Do the job without holding any locks
            job ();
        }

    }

    std::mutex lock_;
    std::condition_variable condVar_;
    bool shutdown_;
    std::queue <std::function <void (void)>> jobs_;
    std::vector <std::thread> threads_;
};
//--------------------------------------------------------------------------Threads end



__global__ void check_ca(char* line, int len, int * ca_p_d) 
{
        char last_c='0'; //The last character seen
	bool found = false;
      	for(int i=((blockIdx.x)*len); i<((blockIdx.x)*len)+len; i++)
        {
                if(last_c=='C' && line[i]=='A')
                {
                        ca_p_d[blockIdx.x]=blockIdx.x;
			found = true;
                        break;
                }
                last_c=line[i];
        }
	if(!found)
		ca_p_d[blockIdx.x]=-1;
}
int main()
{
	string filename;
	ThreadPool p (100);
	// The list in which the file to be processed is present
	ifstream list ("/mnt/pspdata/.init/helices-DB-list");
	while(getline(list, filename))
	{
		//cout<<filename;
		p.doJob (std::bind (process, filename));
		// process(filename);
	}


return 0;
}

void process(string filename)
{
        ifstream myfile ("/mnt/pspdata/.init/helices-DB/"+filename);
        ifstream myfilec ("/mnt/pspdata/.init/helices-DB/"+filename); //copy of that iterator
		ifstream myfiled ("/mnt/pspdata/.init/helices-DB/"+filename);

	int lc = count(std::istreambuf_iterator<char>(myfile),std::istreambuf_iterator<char>(),'\n'); //linecount ie number of lines
	
	string line;    
        getline (myfilec,line);         //linesize
        int ls = line.size()+1;
       
	char whole_file[lc*ls + lc];
        int it=0;
        char c;
        while (myfiled.get(c))          // loop getting single characters
        {    
                whole_file[it]=c;    
                it++;
        }

   
        int ca_p[lc];  // Position of ca, that is line number in which ca is present
	for(int i=0;i<lc;i++)
	{
		ca_p[i]=-1;
	}

	char* lines_d;
	int* ca_p_d;
	hipMalloc((void**)&ca_p_d, lc * sizeof(int)); 
	hipMalloc((void**)&lines_d, (lc*ls+lc)*sizeof(char));
	
	hipMemcpy(lines_d, whole_file , lc*ls*sizeof(char),hipMemcpyHostToDevice);

	check_ca<<<lc,1>>>(lines_d, ls, ca_p_d);
	hipMemcpy(ca_p, ca_p_d, lc*sizeof(int),hipMemcpyDeviceToHost);
	myfile.close();
	
	vector<int> arr_ca;
	for(int i=0; i<lc; i++)
	{
		if(ca_p[i]!=-1)
			arr_ca.push_back(ca_p[i]);
	}

	for(int i=0;i<arr_ca.size();i++)
	{
		if((i+frag_size)<arr_ca.size())
		{
			ofstream outfile ("/mnt/pspdata/.init/profile/helices-DB-threads-100/"+to_string(frag_size)+"_"+to_string(i+1)+"_"+filename);
			int start=arr_ca[i]-1;
			int end=arr_ca[i+frag_size]-2;
			for(int j=start;j<=end;j++)
			{
				for(int k=(j*ls); k<(j*ls)+ls; k++)
				{      
					outfile << whole_file[k];
				}
			}
			outfile.close();
		}
		// if equal to the arr_ca.size() then we will do it differently
		else if((i+frag_size)==arr_ca.size())
		{
			
			ofstream outfile ("/mnt/pspdata/.init/profile/helices-DB-threads-100/"+to_string(frag_size)+"_"+to_string(i+1)+"_"+filename);
			int start=arr_ca[i]-1;
			int end=lc-1;
			for(int j=start;j<=end;j++)
			{
				for(int k=(j*ls); k<(j*ls)+ls; k++)
				{      
					outfile << whole_file[k];
				}
			}
			outfile.close();
		}
	}
}
