
#include <hip/hip_runtime.h>
#include<string>
#include<fstream>
#include<iostream>
#include<algorithm>
#include<vector>
using namespace std;
void process(string,string);
int frag_size=3;

__global__ void check_ca(char* line, int len, int * ca_p_d) 
{
        char last_c='0'; //The last character seen
	bool found = false;
      	for(int i=((blockIdx.x)*len); i<((blockIdx.x)*len)+len; i++)
        {
                if(last_c=='C' && line[i]=='A')
                {
                        ca_p_d[blockIdx.x]=blockIdx.x;
			found = true;
                        break;
                }
                last_c=line[i];
        }
	if(!found)
		ca_p_d[blockIdx.x]=-1;
}


int main()
{
	string filename;
	ifstream list ("/mnt/pspdata/.init/sheet-DB-list");

	double progress = 0;

	while(getline(list, filename))
	{
		//Code for showing the progress bar
		int barWidth = 100;
		cout << "[";
		int pos = barWidth * progress;
		for (int i = 0; i < barWidth; ++i) 
		{
        		if (i < pos) cout << "=";
	        	else if (i == pos) cout << ">";
		        else cout << " ";
		}
		cout << "] " << double(progress * 100.0) <<" "<<filename <<"\t %\r";
		cout.flush();

		progress += 1.0/6959.0;
		cout.flush();
		//Progress bar code ends
		process("/mnt/pspdata/.init/sheet-DB/" + filename,filename);
	}

std::cout << std::endl;
return 0;
}

void process(string filename, string succintfname)
{
        ifstream myfile (filename);
        ifstream myfilec (filename); //copy of that iterator
	ifstream myfiled (filename);

	int lc = count(std::istreambuf_iterator<char>(myfile),std::istreambuf_iterator<char>(),'\n'); //linecount ie number of lines
	
	string line;    
        getline (myfilec,line);         //linesize
        int ls = line.size()+1;
       
	char whole_file[lc*ls + lc];
        int it=0;
        char c;
        while (myfiled.get(c))          // loop getting single characters
        {    
                whole_file[it]=c;    
                it++;
        }

   
        int ca_p[lc];  // Position of ca, that is line number in which ca is present
	for(int i=0;i<lc;i++)
	{
		ca_p[i]=-1;
	}

	char* lines_d;
	int* ca_p_d;
	hipMalloc((void**)&ca_p_d, lc * sizeof(int)); 
	hipMalloc((void**)&lines_d, (lc*ls+lc)*sizeof(char));
	
	hipMemcpy(lines_d, whole_file , lc*ls*sizeof(char),hipMemcpyHostToDevice);

	check_ca<<<lc,1>>>(lines_d, ls, ca_p_d);
	hipMemcpy(ca_p, ca_p_d, lc*sizeof(int),hipMemcpyDeviceToHost);
	myfile.close();
	
	vector<int> arr_ca;
	for(int i=0; i<lc; i++)
	{
		if(ca_p[i]!=-1)
			arr_ca.push_back(ca_p[i]);
	}

	for(int i=0;i<arr_ca.size();i++)
	{
		if((i+frag_size)<=arr_ca.size())
		{
			ofstream outfile ("/mnt/pspdata/.init/frag-sheet/3frag/frag"+to_string(frag_size)+"_"+to_string(i+1)+"_"+succintfname);
			for(int j=i;j<i+frag_size;j++)
			{
				for(int k=(arr_ca[j]*ls); k<(arr_ca[j]*ls)+ls; k++)
				{      
					outfile << whole_file[k];
				}
			}
			outfile.close();
		}
	}
}
