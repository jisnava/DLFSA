
#include <hip/hip_runtime.h>
#include <string>
#include <fstream>
#include<iostream>
//#include <filesystem>
#include<algorithm>
#include<vector>
using namespace std;
int frag_size=19;
string filename = "1RSZ_A_138.pdb";

__global__ void check_prot(char* line, int len, int * ca_p_d) 
{
        char last_c='0'; //The last character seen
	char last_sc='0'; //The last second character seen
	bool found = false;
      	for(int i=((blockIdx.x)*len); i<((blockIdx.x)*len)+len; i++)
        {
                if(last_sc=='G' && last_c=='L' && line[i]=='Y')
                {
                        ca_p_d[blockIdx.x]=blockIdx.x;
			found = true;
                        break;
                }
		last_sc=last_c;
                last_c=line[i];
        }
	if(!found)
		ca_p_d[blockIdx.x]=-1;
}


int main()
{
        ifstream myfile (filename);
        ifstream myfilec (filename); //copy of that iterator
	ifstream myfiled (filename);

	int lc = count(std::istreambuf_iterator<char>(myfile),std::istreambuf_iterator<char>(),'\n'); //linecount ie number of lines
	
	string line;    
        getline (myfilec,line);         //linesize
        int ls = line.size()+1;
       
	char whole_file[lc*ls + lc];
        int it=0;
        char c;
        while (myfiled.get(c))          // loop getting single characters
        {    
                whole_file[it]=c;    
                it++;
        }

   
        int ca_p[lc];  // Position of ca, that is line number in which ca is present
	for(int i=0;i<lc;i++)
	{
		ca_p[i]=-1;
	}

	char* lines_d;
	int* ca_p_d;
	hipMalloc((void**)&ca_p_d, lc * sizeof(int)); 
	hipMalloc((void**)&lines_d, (lc*ls+lc)*sizeof(char));
	
	hipMemcpy(lines_d, whole_file , lc*ls*sizeof(char),hipMemcpyHostToDevice);

	check_prot<<<lc,1>>>(lines_d, ls, ca_p_d);
	hipMemcpy(ca_p, ca_p_d, lc*sizeof(int),hipMemcpyDeviceToHost);
	myfile.close();
	
	vector<int> arr_ca;
	for(int i=0; i<lc; i++)
	{
		if(ca_p[i]!=-1)
			arr_ca.push_back(ca_p[i]);
	}	

	int mod3=0;//Is actually 4
	ofstream outfile ("GLY_"+filename);

	for(int i=0;i<arr_ca.size();i++)
	{
		if(mod3==0)
			outfile << "\n";

		for(int k=(arr_ca[i]*ls); k<(arr_ca[i]*ls)+ls; k++)
		{      
			outfile << whole_file[k];
		}
		mod3=(mod3+1)%4;
	}
	outfile.close();

return 0;
}
