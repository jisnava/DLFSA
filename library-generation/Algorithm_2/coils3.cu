
#include <hip/hip_runtime.h>
#include<string>
#include<fstream>
#include<iostream>
#include<algorithm>
#include<vector>
using namespace std;
int frag_size=3;
void process(string);

__global__ void check_ca(char* line, int len, int * ca_p_d) 
{
        char last_c='0'; //The last character seen
	bool found = false;
      	for(int i=((blockIdx.x)*len); i<((blockIdx.x)*len)+len; i++)
        {
                if(last_c=='C' && line[i]=='A')
                {
                        ca_p_d[blockIdx.x]=blockIdx.x;
			found = true;
                        break;
                }
                last_c=line[i];
        }
	if(!found)
		ca_p_d[blockIdx.x]=-1;
}
int main()
{
	string filename;
	ifstream list ("/mnt/pspdata/.init/coils-DB-list");
	while(getline(list, filename))
	{
		cout<<filename;
		process(filename);
	}


return 0;
}

void process(string filename)
{
        ifstream myfile ("/mnt/pspdata/.init/coils-DB/"+filename);
        ifstream myfilec ("/mnt/pspdata/.init/coils-DB/"+filename); //copy of that iterator
	ifstream myfiled ("/mnt/pspdata/.init/coils-DB/"+filename);

	int lc = count(std::istreambuf_iterator<char>(myfile),std::istreambuf_iterator<char>(),'\n'); //linecount ie number of lines
	
	string line;    
        getline (myfilec,line);         //linesize
        int ls = line.size()+1;
       
	char whole_file[lc*ls + lc];
        int it=0;
        char c;
        while (myfiled.get(c))          // loop getting single characters
        {    
                whole_file[it]=c;    
                it++;
        }

   
        int ca_p[lc];  // Position of ca, that is line number in which ca is present
	for(int i=0;i<lc;i++)
	{
		ca_p[i]=-1;
	}

	char* lines_d;
	int* ca_p_d;
	hipMalloc((void**)&ca_p_d, lc * sizeof(int)); 
	hipMalloc((void**)&lines_d, (lc*ls+lc)*sizeof(char));
	
	hipMemcpy(lines_d, whole_file , lc*ls*sizeof(char),hipMemcpyHostToDevice);

	check_ca<<<lc,1>>>(lines_d, ls, ca_p_d);
	hipMemcpy(ca_p, ca_p_d, lc*sizeof(int),hipMemcpyDeviceToHost);
	myfile.close();
	
	vector<int> arr_ca;
	for(int i=0; i<lc; i++)
	{
		if(ca_p[i]!=-1)
			arr_ca.push_back(ca_p[i]);
	}

	for(int i=0;i<arr_ca.size();i++)
	{
		if((i+frag_size)<arr_ca.size())
		{
			ofstream outfile ("/mnt/pspdata/.init/frag-coils/3frag/frag"+to_string(frag_size)+"_"+to_string(i+1)+"_"+filename);
			int start=arr_ca[i]-1;
			int end=arr_ca[i+frag_size]-2;
			for(int j=start;j<=end;j++)
			{
				for(int k=(j*ls); k<(j*ls)+ls; k++)
				{      
					outfile << whole_file[k];
				}
			}
			outfile.close();
		}
		// if equal to the arr_ca.size() then we will do it differently
		else if((i+frag_size)==arr_ca.size())
		{
			
			ofstream outfile ("/mnt/pspdata/.init/frag-coils/3frag/frag"+to_string(frag_size)+"_"+to_string(i+1)+"_"+filename);
			int start=arr_ca[i]-1;
			int end=lc-1;
			for(int j=start;j<=end;j++)
			{
				for(int k=(j*ls); k<(j*ls)+ls; k++)
				{      
					outfile << whole_file[k];
				}
			}
			outfile.close();
		}
	}
}